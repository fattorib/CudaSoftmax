#include "common.hpp"
#include "kernels/softmax.cuh"
#include "softmax_cpu.hpp"
#include <algorithm>
#include <cassert>
#include <cmath>
#include <iostream>
#include <vector>

#define CHECK_TRUE(v) (assert(v == true))

int main(void) {
  const long numRows = 4096;

  const long numCols = 16384;

  long warmup_iters, benchmark_iters;

#ifdef PROFILE
  warmup_iters = 1;
  benchmark_iters = 1;

#else
  warmup_iters = 25;
  benchmark_iters = 1000;

#endif

  float *d_in, *d_out;

  // make host-side arrays
  std::vector<float> in(numRows * numCols);
  std::vector<float> out(numRows * numCols);

  fill_matrix(in);

  // allocate memory for GPU arrays
  hipMalloc(&d_in, in.size() * sizeof(float));
  hipMalloc(&d_out, out.size() * sizeof(float));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // copy from host to GPU
  hipMemcpy(d_in, in.data(), in.size() * sizeof(float),
             hipMemcpyHostToDevice);

  const int numThread = 128; // makes sense to think about this as # of warps
                             // since we are doing warp reductions

  dim3 blockDim(numThread);

  assert(numCols % (4 * numThread) == 0);

  // launch one threadblock per row
  dim3 gridDim(numRows);

  int bytesBlock = sizeof(float) * numCols;

  hipFuncSetAttribute(reinterpret_cast<const void*>(softmax_kernel<numCols), numThread>,
                       hipFuncAttributeMaxDynamicSharedMemorySize, bytesBlock);

  std::cout << "Warmup started" << std::endl;
  for (int i = 0; i < warmup_iters; i++) {
    softmax_kernel<numCols, numThread>
        <<<gridDim, blockDim, bytesBlock>>>(d_in, d_out);
  }

  hipDeviceSynchronize();
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }

  std::cout << "Benchmark started" << std::endl;
  hipEventRecord(start);
  for (int i = 0; i < benchmark_iters; i++) {
    softmax_kernel<numCols, numThread>
        <<<gridDim, blockDim, bytesBlock>>>(d_in, d_out);
  }

  hipEventRecord(stop);

  hipMemcpy(out.data(), d_out, out.size() * sizeof(float),
             hipMemcpyDeviceToHost);
  hipEventSynchronize(stop);

  float milliseconds;
  hipEventElapsedTime(&milliseconds, start, stop);

  double gb = benchmark_iters * 2 * sizeof(float) * numRows * numCols * 1e-9;
  double elapsed_time = double(milliseconds) * 1e-3;

  printf("Total elapsed time: (%7.6f) s, performance: (%7.1f) GB/s, memory "
         "reads & writes (GB): (%6.1lf) \n\n",
         elapsed_time, (gb) / elapsed_time, gb);

  std::vector<float> cpuOut(numRows * numCols);

  softmaxCpu<numRows, numCols>(in, cpuOut);

  // compute errors
  printf("Error checking:\n");
  printf("Relative Error (%7.8f) \n", relative_error(cpuOut, out));
}